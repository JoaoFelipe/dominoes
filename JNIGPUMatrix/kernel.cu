#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <algorithm>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <stdio.h>

#define N_THREADS_X 8
#define N_THREADS_Y 8

__global__ void K_CheckNonZerosInCol(float *raw, int rows, int cols, int *nonZeros){
	
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (idx < rows){
		
		for (int i = 0; i < cols; i++){
			if (raw[idx * cols + i] > 0){
				nonZeros[idx] = 1;
			}
		}
	}
	
}

__global__ void K_Mean(float* mat_sum_depths, float *out_mean, int rows, int cols, float divide_by){
	
	float sum_depths  = 0;

	int columIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (columIdx < cols){
		
		
		for (int i = 0; i < rows; i++){
			sum_depths += mat_sum_depths[columIdx + (i * cols)];
		}
		out_mean[columIdx] = sum_depths / divide_by ;
	}
}

__global__ void K_Variance(float* layer, float* mean, int rows, int cols){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if ((idx < cols) && (idy < rows)){
		float meanv = mean[idx];
		float value = meanv - layer[idy * cols + idx];

		layer[idy * cols + idx] = value * value;
	}
}

__global__ void K_StandardDeviation(float* mat_sum_depths, float *out_sd, int rows, int cols, int depths){

	float sum_depths  = 0;

	int columIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (columIdx < cols){
		for (int i = 0; i < rows; i++){
			sum_depths += mat_sum_depths[columIdx + (i * cols)];
		}
		out_sd[columIdx] = sqrt(sum_depths / (rows  * depths));
	}
}

__global__ void AddKernel(float* _mat1, float *_mat2, float *_res, 
	int rows1, int cols, int cols2){
    
    int idX = blockDim.x * blockIdx.x + threadIdx.x;
    int idY = blockDim.y * blockIdx.y + threadIdx.y;
     
    
    int id = idY * cols2 + idX;
    
    if (id < rows1 * cols2){
    
    	_res[id] = 0;
    
    	int mat1_row = idY * cols;

    	for ( int i = 0; i < cols; i++){
    		int mat2_col = i * cols2 + idX;
    		_res[id] += _mat1[mat1_row + i] * _mat2[mat2_col];
    	}
    }
}

__global__ void StandardScoreKernel(float* _mat, int rows, int cols,
	float* meanSD, float *res){
    
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (idx < cols){
    
    	for (int i = 0; i < rows; i++){
    		int idxElement = i * cols + idx;    		    						
			float _mean = meanSD[idx];
			float _sd = meanSD[cols + idx];
			
			res[idxElement] = (_mat[idxElement] - _mean) / _sd;
		}
    }
}

extern "C" {
    void g_MatMul(float* _mat1, float *_mat2, float *_res, 
		int rows1, int cols, int cols2){
    
    	float *d_mat1;
    	float *d_mat2;
    	float *d_result;
    	
    	checkCudaErrors(hipMalloc(&d_mat1, sizeof(float) * rows1 * cols));
    	checkCudaErrors(hipMemcpy(d_mat1, _mat1, sizeof(float) * rows1 * cols, hipMemcpyHostToDevice));
    	
    	checkCudaErrors(hipMalloc(&d_mat2, sizeof(float) * cols * cols2));
    	checkCudaErrors(hipMemcpy(d_mat2, _mat2, sizeof(float) * cols * cols2, hipMemcpyHostToDevice));
    	
    	checkCudaErrors(hipMalloc(&d_result, sizeof(float) * rows1 * cols2));
        
       
        dim3 blockDim(N_THREADS_X, N_THREADS_Y, 1);
        dim3 gridDim( ceil((float)cols2/N_THREADS_X), ceil((float)rows1/N_THREADS_Y), 1);
        printf("gridx: %d, gridy: %d\n", gridDim.x, gridDim.y);

        AddKernel<<<gridDim, blockDim>>>(d_mat1, d_mat2, d_result, rows1, cols, cols2);
        
            
        checkCudaErrors(hipDeviceSynchronize());
    	checkCudaErrors(hipMemcpy(_res, d_result, sizeof(float) * rows1 * cols2, hipMemcpyDeviceToHost));
    	checkCudaErrors(hipFree(d_mat1));
    	checkCudaErrors(hipFree(d_mat2));
    	checkCudaErrors(hipFree(d_result));
    }
    
    
    void g_StandardDeviation(float* mat, int rows, int cols, 
		float* meanSD, float* result){
		
		float *d_mat;
    	float *d_meanSD;
    	float *d_result;
    	
    	checkCudaErrors(hipMalloc(&d_mat, sizeof(float) * rows * cols));
    	checkCudaErrors(hipMemcpy(d_mat, mat, sizeof(float) * rows * cols, hipMemcpyHostToDevice));
    	
    	checkCudaErrors(hipMalloc(&d_meanSD, sizeof(float) * 2 * cols));
    	checkCudaErrors(hipMemcpy(d_meanSD, meanSD, sizeof(float) * 2 * cols, hipMemcpyHostToDevice));
    	
    	checkCudaErrors(hipMalloc(&d_result, sizeof(float) * rows * cols));
    	
    	dim3 blockDim(N_THREADS_X * N_THREADS_Y, 1, 1);
        dim3 gridDim(ceil((float)cols/(N_THREADS_X *  N_THREADS_Y)), 1, 1);
        
        StandardScoreKernel<<<gridDim, blockDim>>>(d_mat, rows, cols, d_meanSD, d_result);
        
        checkCudaErrors(hipMemcpy(result, d_result, sizeof(float) * rows * cols, hipMemcpyDeviceToHost));
    	checkCudaErrors(hipFree(d_mat));
    	checkCudaErrors(hipFree(d_meanSD));
    	checkCudaErrors(hipFree(d_result));
	}
    
    
    void g_MeanSD(int rows, int cols, int depth, float *h_data, float *result, bool considerZeros){
    	//hipSetDevice(0);

    	// Generate the keys array
    	int h_layer_keys[rows * cols];
    	for (int i = 0; i < rows; i++){
    		
    		for (int j = 0; j < cols; j++){
    			h_layer_keys[i * cols + j] = j;
    		}
    	}


    	float *d_raw, *d_val_res, *d_mean_sd, *d_sum_depths;
    	int *d_kraw, *d_keys_res, *d_nonZeros;
    	checkCudaErrors(hipMalloc((void**) &d_raw, sizeof(float) * rows * cols));
    	checkCudaErrors(hipMalloc((void**) &d_kraw, sizeof(int) * rows * cols));
    	checkCudaErrors(hipMalloc((void**) &d_val_res, sizeof(float) * rows * cols));	
    	checkCudaErrors(hipMalloc((void**) &d_keys_res, sizeof(int) * rows * cols));
    	checkCudaErrors(hipMalloc((void**) &d_mean_sd, sizeof(float) * cols * 2));
    	checkCudaErrors(hipMalloc((void**) &d_sum_depths, sizeof(float) * depth * cols));
    	checkCudaErrors(hipMemset(d_val_res, 0, sizeof(float) * rows * cols));
    	checkCudaErrors(hipMemset(d_keys_res, 0, sizeof(int) * rows * cols));
    	
    	if (!considerZeros){
    		checkCudaErrors(hipMalloc((void**) &d_nonZeros, sizeof(int) * rows));
    		checkCudaErrors(hipMemset(d_nonZeros, 0, sizeof(int) * rows));
    	}
    		
    		


    	thrust::device_ptr<float> dev_ptr(d_raw);
    	thrust::device_ptr<int> dev_ptr_k(d_kraw);
    	thrust::device_ptr<int> dev_ptr_k_res(d_keys_res);
    	thrust::device_ptr<float> dev_ptr_v_res(d_val_res);


    	for (int i = 0; i < depth; i++){
    		checkCudaErrors(hipMemcpy(d_raw, &h_data[i * rows * cols], 
    				sizeof(float) * rows * cols, hipMemcpyHostToDevice));
    		checkCudaErrors(hipMemcpy(d_kraw, h_layer_keys, 
    				sizeof(int) * rows * cols, hipMemcpyHostToDevice));
    		
    		// check non zeros
    		if (!considerZeros){
    			dim3 blockDim_nz(256, 1, 1);
    			dim3 gridDim_nz( ceil((float)rows/256), 1, 1);
    			K_CheckNonZerosInCol<<<gridDim_nz, blockDim_nz>>>(d_raw, rows, cols, d_nonZeros);
    		}
        	       	
    		thrust::sort_by_key(dev_ptr_k, dev_ptr_k + (rows * cols), dev_ptr);
    		hipDeviceSynchronize();
    		thrust::reduce_by_key(dev_ptr_k, dev_ptr_k+(rows * cols), dev_ptr, dev_ptr_k_res, dev_ptr_v_res);
    		hipDeviceSynchronize();
    		checkCudaErrors(hipMemcpy(&d_sum_depths[i * cols], d_val_res, 
    				sizeof(float) * cols, hipMemcpyDeviceToDevice));
    		
    	}
    	
    	float _result[cols*depth];
    	hipMemcpy(_result, d_mean_sd, sizeof(float) * cols * depth, hipMemcpyDeviceToHost);

    	for (int i = 0; i < cols*depth; i++)
    		fprintf(stderr, "mean: %f\n", _result[i]);

    	// Calculate the mean
    	int divide_by = rows * depth;
    	
    	if (!considerZeros){
    		thrust::device_ptr<int> dev_ptr_nonZeros(d_nonZeros);
    		divide_by = thrust::reduce(dev_ptr_nonZeros, dev_ptr_nonZeros + rows) * depth;
    		fprintf(stderr, "dividby: %d\n", divide_by);
    	}
    	hipDeviceSynchronize();
    	
    	dim3 blockDim_m(256, 1, 1);
    	dim3 gridDim_m( ceil((float)cols/256), 1, 1);
    	K_Mean<<<gridDim_m, blockDim_m>>>(d_sum_depths, d_mean_sd, rows, cols, (float)divide_by);
    	hipDeviceSynchronize();
    	
    	
    	/*float _result[cols];
    	hipMemcpy(_result, d_mean_sd, sizeof(float) * cols, hipMemcpyDeviceToHost);

    	for (int i = 0; i < cols; i++)
    		fprintf(stderr, "mean: %f\n", _result[i]);*/

    	// Variance
    	/*dim3 blockDim_v(N_THREADS_X, N_THREADS_Y);
    	dim3 gridDim_v( ceil((float)cols/N_THREADS_X),ceil((float)rows/N_THREADS_Y));

    	for (int i = 0; i < depth; i++){		
    		checkCudaErrors(hipMemcpy(d_raw, &h_data[i * rows * cols], 
    				sizeof(float) * rows * cols, hipMemcpyHostToDevice));
    		checkCudaErrors(hipMemcpy(d_kraw, h_layer_keys, 
    				sizeof(int) * rows * cols, hipMemcpyHostToDevice));
        	K_Variance<<<gridDim_v, blockDim_v>>>(d_raw, d_mean_sd, rows, cols);

    		thrust::sort_by_key(dev_ptr_k, dev_ptr_k + (rows * cols), dev_ptr);
    		thrust::reduce_by_key(dev_ptr_k, dev_ptr_k+(rows * cols), dev_ptr, dev_ptr_k_res, dev_ptr_v_res);
    		checkCudaErrors(hipMemcpy(&d_sum_depths[i * cols], d_val_res, 
    				sizeof(float) * cols, hipMemcpyDeviceToDevice));
    		hipDeviceSynchronize();
    	}
    	
    	dim3 blockDim_s(256, 1, 1);
    	dim3 gridDim_s( ceil((float)cols/256), 1, 1);
    	K_StandardDeviation<<<gridDim_s, blockDim_s>>>(d_sum_depths, &d_mean_sd[cols], rows, cols, depth);
    	hipDeviceSynchronize();*/

    	
    	checkCudaErrors(hipMemcpy(result, d_mean_sd, 
    			sizeof(float) * cols * 2, hipMemcpyDeviceToHost));


    	if (!considerZeros){
    		hipFree(d_nonZeros);
    	}
    	hipFree(d_sum_depths);
    	hipFree(d_mean_sd);   	
    	hipFree(d_raw);
    	hipFree(d_kraw);
    	hipFree(d_val_res);
    	hipFree(d_keys_res);
    	
    	
    }
    
    
    void g_SparseMultiplication(){
    
    }
    
} 